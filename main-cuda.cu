#include "hip/hip_runtime.h"




//
// Created by Filippos Kasioulis on 25/01/2019.
//

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>


typedef  struct  {
    float x;
    float y;
    float z;

}Point;




__global__ void knn_search(Point* allcPoints,Point* allqPoints,int* perBlockcPoints,int* perBlockqPoints,int* startingPointc,int* startingPointq,Point* knn,float* knn_dist){


    __shared__ Point shrMem[2028];
    int blockId=blockIdx.x+blockIdx.y*gridDim.x+blockIdx.z*gridDim.x*gridDim.y;

    Point qpoint;
    Point cpoint;
    int c,q;
    int number_qpoints=perBlockqPoints[blockId];
    int number_cpoints=perBlockcPoints[blockId];
    int i;

for(i=threadIdx.x;i<number_cpoints;i+=blockDim.x) {
    c = i + startingPointc[blockId];
    shrMem[i] = allcPoints[c];
}
    __syncthreads();

 for(i=threadIdx.x;i<number_qpoints;i+=blocDim.x){

        q = i + startingPointq[blockId];
        qpoint = allqPoints[q];
        float tempDist = 0;
        float minCanDist = 999;
        Point minCanPoint;
        for (int c = 0; c < number_cpoints; c++) {

            tempDist = pow((shrMem[c].x - qpoint.x), 2) + pow((shrMem[c].y - qpoint.y), 2) +
                       pow((shrMem[c].z - qpoint.z), 2);
            tempDist = sqrt(tempDist);


            if (tempDist < minCanDist) {
                minCanDist = tempDist;
                minCanPoint = shrMem[i];

            }
        }

        knn[q] = minCanPoint;
        knn_dist[q] = minCanDist;
    }

    //float minBounds = 999;
    //float tempDistancex = qpoints[q].x-(block_length * blockId.x);
    //if ((tempDistancex > block_length - tempDistancex)&&(blockId.x<gridDim.x-1))
    //  tempDistancex = block_length - tempDistancex;
    //min_from_bounds = tempDistancex;
    //float tempDistancey = qpoints[q].y-(block_length * blockId.y);
    //if ((tempDistancey > block_length - tempDistancey)&&(blockId.y<gridDim.y-1))
    //   tempDistancey = block_length - tempDistancey;
    //if (tempDistancey < min_from_bounds)
    //min_from_bounds = tempDistancey;

    // float tempDistancez = qpoints[q].z-(block_length * blockDim.z);
    //if ((tempDistancez > block_length - tempDistancez)&&(blockId.z<gridDim.z-1))
    //tempDistancez = block_length - tempDistancez;
    // if (tempDistancez < min_from_bounds)
    // min_from_bounds = tempDistancez;


    int neighbors[27];
    int size = 0;
    for (int i = -1; i < 2; i++) {
        for (int j = -1; j < 2; j++) {
            for (int k = -1; k < 2; k++) {
                if (!(i == 0 & j == 0 & k == 0)) {
                    int tempx = blockIdx.x + i;
                    int tempy = blockIdx.y + j;
                    int tempz = blockIdx.z + k;
                    if (!(tempx < 0 | tempy < 0 | tempz < 0 | tempx >= gridDim.x | tempz >= gridDim.y |
                          tempy >= gridDim.z)) {
                        neighbors[size] = tempx + tempy * gridDim.x + tempz * gridDim.x * gridDim.y;
                        size++;
                    }
                }
            }
        }
    }




    float minNeigDist=999;
    Point minNeig;

    for (int k=0;k<size;k++){

        int neighborId=neighbors[k];
        number_cpoints=perBlockcPoints[neighborId];
        number_qpoints=perBlockqPoints[neighborId];

       for(i=threadIdx.x;i<number_cpoints;i=i+blockDim.x){
            c=i+startingPointc[neighborId];
            shrMem[i]=allcPoints[c];
        }
        __syncthreads();
        for(i=threadIdx.x;i<number_qpoints;i+blockDim.x){
            q=i+startingPointq[neighborId];
            qpoint=allqPoints[q];
            float tempDist;
            for(int k=0;k<number_cpoints;k++){

                tempDist = pow((shrMem[k].x - qpoint.x), 2) + pow((shrMem[k].y - qpoint.y), 2) +
                           pow((shrMem[k].z - qpoint.z), 2);
                tempDist = sqrt(tempDist);

                if (tempDist < minNeigDist) {
                    minNeigDist = tempDist;
                    minNeig =shrMem[k];
                }
            }
            if (minNeigDist < knn_dist[q]) {
                knn_dist[q] = minNeigDist;
                knn[q] = minNeig;
        }
     }
    }
}


void PrintPoints(Point *arr,int number){
    for (int i = 0; i < number; i++) {

        printf("Point %d: x=%f y=%f z=%f \n", i,arr[i].x, arr[i].y, arr[i].z);
    }
}

void PrintKnn( Point *arr, Point *arr2,int number ) {
    for (int i = 0; i < number; i++) {

        printf("Nearest to Point %d: x=%f y=%f z=%f--->  x=%f y=%f z=%f\n", i,arr[i].x, arr[i].y, arr[i].z,arr2[i].x, arr2[i].y, arr2[i].z);

    }
}

void generatePoints(Point* arr ,int number,int seed){
    srand(seed);// randomize seed
    for (int i=0;i<number;i++){
        arr[i].x=((float)rand()/(float)RAND_MAX);
        arr[i].y=((float)rand()/(float)RAND_MAX);
        arr[i].z=((float)rand()/(float)RAND_MAX);

    }

}

float distanceOfPoints(  Point p1, Point p2){
    float dist=0;
    dist=pow(p1.x-p2.x,2)+pow(p1.y-p2.y,2)+pow(p1.z-p2.z,2);
    dist=sqrt(dist);
    return dist;
}


void putPointsBlocks(int *pointsToBlock, Point* allPoints,int* perBlockPoints,int numberOfPoints,int gridDim,float blockLength) {
    int blockId=0;
    for (int i=0; i<numberOfPoints;i++){

        blockId=floor(allPoints[i].x/blockLength)+floor(allPoints[i].y/blockLength)*gridDim+floor(allPoints[i].z/blockLength)*gridDim*gridDim;
        pointsToBlock[i]=blockId;
        perBlockPoints[blockId]++;

    }
}



void arrangePointsbyblock(int* pointsToBlock, Point* allPoints, Point* newPoints,int numberOfPoints,int numberOfBlocks,int* perBlockPoints,int* startingPoint){
    int k=0;
    int sum=0;
    for(int j=0;j<numberOfBlocks;j++){
        for (int i=0;i<numberOfPoints;i++) {
            if (pointsToBlock[i] == j) {
                newPoints[k] = allPoints[i];
                k++;

            }
        }
        startingPoint[j]=sum;
        sum=sum+perBlockPoints[j];
    }
}



int main(int argc,char** argv){

    int numberOfcPoints = 22;
    numberOfcPoints = pow(2, numberOfcPoints);
    int dimOfGrid = 6;
    dimOfGrid = pow(2, dimOfGrid);
    int numberOfBlocks = pow(dimOfGrid, 3);
    int numberOfqpoints = numberOfcPoints;
    Point* cpoints = (Point*)malloc(numberOfcPoints * sizeof( Point));
    Point* qpoints = (Point*)malloc(numberOfqpoints * sizeof( Point));

    Point* knn = (Point*)malloc(numberOfqpoints * sizeof( Point));
    Point* arrangecpoints=(Point*)malloc(numberOfcPoints*sizeof( Point));
    Point* arrangeqpoints=(Point*)malloc(numberOfqpoints*sizeof(Point));
    int* pointsctoblock = (int*)malloc(numberOfcPoints * sizeof(int));
    int* pointsqtoblock = (int*)malloc(numberOfqpoints * sizeof(int));
    int* perblockcpoints = (int*)malloc(numberOfBlocks * sizeof(int));
    int* perblockqpoints = (int*)malloc(numberOfBlocks * sizeof(int));
    int* startingpoint_c=(int*)malloc(numberOfBlocks*sizeof(int));
    int* startingpoint_q=(int*)malloc(numberOfBlocks*sizeof(int));
    float* knn_Dist=(float*)malloc(numberOfqpoints*sizeof(float));

    generatePoints(cpoints,numberOfcPoints,1);
    generatePoints(qpoints,numberOfqpoints,2);
    //printf("-------------C  POINTS-----------\n");
    //PrintPoints(cpoints,numberOfcPoints);
    //printf("-------------Q POINTS------------\n");
    //PrintPoints(qpoints,numberOfqpoints);
    struct timeval start_t,end_t;
    double par_time;
    gettimeofday(&start_t,NULL);
    float block_length = ((float) 1) / ((float) dimOfGrid);

    //call function for fragmentation
    putPointsBlocks(pointsctoblock, cpoints, perblockcpoints, numberOfcPoints, dimOfGrid, block_length);
    putPointsBlocks(pointsqtoblock, qpoints, perblockqpoints, numberOfqpoints, dimOfGrid, block_length);
    arrangePointsbyblock(pointsctoblock,cpoints,arrangecpoints,numberOfcPoints,numberOfBlocks,perblockcpoints,startingpoint_c);
    arrangePointsbyblock(pointsqtoblock,qpoints,arrangeqpoints,numberOfqpoints,numberOfBlocks,perblockqpoints,startingpoint_q);
    free(pointsctoblock);
    free(pointsqtoblock);
    hipError_t err;
    Point* arrangecpointsDev;
    err=hipMalloc(&arrangecpointsDev,numberOfcPoints*sizeof(Point));
    if (err != hipSuccess)                             // `hipSuccess` is provided by CUDA.
    {
        printf("Error: %s\n", hipGetErrorString(err)); // `hipGetErrorString` is provided by CUDA.
    }
    hipMemcpy(arrangecpointsDev,arrangecpoints,numberOfcPoints*sizeof(Point),hipMemcpyHostToDevice);
    Point* arrangeqpointsDev;
    err=hipMalloc(&arrangeqpointsDev,numberOfqpoints*sizeof(Point));
    if (err != hipSuccess)                             // `hipSuccess` is provided by CUDA.
    {
        printf("Error: %s\n", hipGetErrorString(err)); // `hipGetErrorString` is provided by CUDA.
    }
    hipMemcpy(arrangeqpointsDev,arrangeqpoints,numberOfqpoints*sizeof(Point),hipMemcpyHostToDevice);

    int* startingpointDev_c;
    err=hipMalloc(&startingpointDev_c,numberOfBlocks*sizeof(int));
    if (err != hipSuccess)                             // `hipSuccess` is provided by CUDA.
    {
        printf("Error: %s\n", hipGetErrorString(err)); // `hipGetErrorString` is provided by CUDA.
    }
    hipMemcpy(startingpointDev_c,startingpoint_c,numberOfBlocks*sizeof(int),hipMemcpyHostToDevice);
    int* startingpointDev_q;

    err=hipMalloc(&startingpointDev_q,numberOfBlocks*sizeof(int));
    if (err != hipSuccess)                             // `hipSuccess` is provided by CUDA.
    {
        printf("Error: %s\n", hipGetErrorString(err)); // `hipGetErrorString` is provided by CUDA.
    }
    hipMemcpy(startingpointDev_q,startingpoint_q,numberOfBlocks*sizeof(int),hipMemcpyHostToDevice);

    int* perblockcpointsDev;
    err=hipMalloc(&perblockcpointsDev,numberOfBlocks*sizeof(int));
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(perblockcpointsDev,perblockcpoints,numberOfBlocks*sizeof(int),hipMemcpyHostToDevice);
    int *perblockqpointsDev;
    err=hipMalloc(&perblockqpointsDev,numberOfBlocks*sizeof(int));
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(perblockqpointsDev,perblockqpoints,numberOfBlocks*sizeof(int),hipMemcpyHostToDevice);
    Point* knn_Dev=NULL;
    err=hipMalloc(&knn_Dev,numberOfqpoints*sizeof(Point));
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    float* knnDist_Dev=NULL;
    err=hipMalloc(&knnDist_Dev,numberOfqpoints*sizeof(float));
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }


    knn_search<<<dim3(dimOfGrid,dimOfGrid,dimOfGrid),1024>>>(arrangecpointsDev,arrangeqpointsDev,perblockcpointsDev,perblockqpointsDev,startingpointDev_c,startingpointDev_q,knn_Dev,knnDist_Dev);
    gettimeofday(&end_t,NULL);
    par_time = (double)((end_t.tv_usec - start_t.tv_usec)/1.0e6
                        + end_t.tv_sec - start_t.tv_sec);

    hipMemcpy(knn_Dist,knnDist_Dev,numberOfqpoints*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(knn,knn_Dev,numberOfqpoints*sizeof(Point),hipMemcpyDeviceToHost);
    printf("Time to )
    //PrintKnn(arrangeqpoints,knn,numberOfqpoints);


    free(cpoints);
    free(qpoints);
    free(arrangecpoints);
    free(arrangeqpoints);
    free(perblockcpoints);
    free(perblockqpoints);
    free(startingpoint_c);
    free(startingpoint_q);
    free(knn);
    free(knn_Dist);

    hipDeviceReset();

    return 0;
}






