#include "hip/hip_runtime.h"
//
// Created by Filippos Kasioulis on 25/01/2019.
//

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>


typedef  struct  {
    float x;
    float y;
    float z;

}Point;




__global__ void knn(float* allcPoints,float* allqPoints,int * perBlockcPoints,int* perBlockqPoints,int* startingPointc,int* startingPointq,int* knn,int* knn){

    __shared__ Point shrMem[1024];
    int blockId=blockIdx.x+blockIdx.y*gridDim.x+blockIdx.z*gridDim.x*gridDim.y;

    Point qpoint;
    Point cpoint;
    int c,q;
    int number_qpoints=perBlockqPoints[blockId];
    int number_cpoints=perBlcokcPoints[blockId];
    int i=threadId.x;


    if(i<number_cpoints){
         c=i+startingPoint[blockId];
         shrMem[threadId.x]=allcPoints[c];

    }
    __syncthreads();

    if(i<number_qpoints) {

        q = i + startingPointq[blockId];
        qpoint = allqpoints[q];
        float tempDist = 0;
        float minCanDist=999;
        struct Point minCanPoint;
        for (int c = 0; c < number_cpoints; c++) {

            tempDist = pow((shrMem[c].x - qpoint.x), 2) + pow((shrMem[c].y - qpoint.y), 2) +
                       pow((shrMem[c].z - qpoint.z), 2);
            tempDist = sqrt(tempDist);


            if (tempDist < minCanDis) {
                minCanDis = tempDist;
                minCanPoint = shrMem[c];

            }
        }

        knn[q]=minCanPoint;
        knn_dist[q]=minCanDist;
    }

    //float minBounds = 999;
    //float tempDistancex = qpoints[q].x-(block_length * blockId.x);
    //if ((tempDistancex > block_length - tempDistancex)&&(blockId.x<gridDim.x-1))
      //  tempDistancex = block_length - tempDistancex;
    //min_from_bounds = tempDistancex;
    //float tempDistancey = qpoints[q].y-(block_length * blockId.y);
    //if ((tempDistancey > block_length - tempDistancey)&&(blockId.y<gridDim.y-1))
     //   tempDistancey = block_length - tempDistancey;
    //if (tempDistancey < min_from_bounds)
        //min_from_bounds = tempDistancey;

   // float tempDistancez = qpoints[q].z-(block_length * blockDim.z);
    //if ((tempDistancez > block_length - tempDistancez)&&(blockId.z<gridDim.z-1))
        //tempDistancez = block_length - tempDistancez;
   // if (tempDistancez < min_from_bounds)
       // min_from_bounds = tempDistancez;


        int neighbors[27];
        int size=0;
        for (int i = -1; i < 2; i++) {
            for (int j = -1; j < 2; j++) {
                for (int k = -1; k < 2; k++) {
                    if (!(i == 0 & j == 0 & k == 0)) {
                        int tempx = blockIdx.x+i;
                        int tempy = blockIdx.y+j;
                        int tempz = blockIdx.z+k;
                        if (!(tempx < 0 | tempy < 0 | tempz < 0 | tempx >= gridDim | tempz >= gridDim | tempy >= gridDim)) {
                            neighbors[size]=tempx+tempy*gridDim+tempz*gridDim*gridDim;
                            size++;
                        }
                    }
                }
            }
        }
     float minNeigDist=999;
     Point minNeig;
     for (int k=0;k<size;k++){

     int neighborId=neighbors[k];
     number_cpoints=perBlockcPoints[neighborId];
     number_qpoints=perBlockqPoints[neighborId];

     if(i<number_cpoints]){
         c=i+startingPointc[neighborId];
         shrMem[i]=allcPoints[c];
     }
     __syncthreads();
     if(i<number_qpoints){
      q=i+startingPointq[neighborId];
      qpoint=allqPoints[q];
      for(int c=0;c<number_qpoints;c++){

          tempDist = pow((shrMem[c].x - qpoint.x), 2) + pow((shrMem[c].y - qpoint.y), 2) +
                     pow((shrMem[c].z - qpoint.z), 2);
          tempDist = sqrt(tempDist);

          if (tempDist < minNeigDist) {
              minNeigDis = tempDist;
              minNeig =shrMem[c];
          }
      }
     }
    }
    if(i<number_qpoints) {
        if (minNeigDist < knn[q]) {
            knn[q] = minNeigDist;
            knn[q] = minNeig;
        }
    }
 }


void PrintPoints(struct Point *arr,int number){
    for (int i = 0; i < number; i++) {

        printf("Point %d: x=%f y=%f z=%f \n", i,arr[i].x, arr[i].y, arr[i].z);
    }
}

void PrintKnn(struct Point *arr,struct Point *arr2,int number ) {
    for (int i = 0; i < number; i++) {

        printf("Nearest to Point %d: x=%f y=%f z=%f--->  x=%f y=%f z=%f\n", i,arr[i].x, arr[i].y, arr[i].z,arr2[i].x, arr2[i].y, arr2[i].z);

    }
}

void generatePoints( struct Point* arr ,int number,int seed){
    srand(seed);// randomize seed
    for (int i=0;i<number;i++){
        arr[i].x=((float)rand()/(float)RAND_MAX);
        arr[i].y=((float)rand()/(float)RAND_MAX);
        arr[i].z=((float)rand()/(float)RAND_MAX);

    }

}

float distanceOfPoints( struct Point p1,struct Point p2){
    float dist=0;
    dist=pow(p1.x-p2.x,2)+pow(p1.y-p2.y,2)+pow(p1.z-p2.z,2);
    dist=sqrt(dist);
    return dist;
}


void putPointsBlocks(int *pointsToBlock,struct Point* allPoints,int* perBlockPoints,int numberOfPoints,int gridDim,float blockLength,struct Point2* pointsToBlockDim) {
    int blockId=0;
    for (int i=0; i<numberOfPoints;i++){

        blockId=floor(allPoints[i].x/blockLength)+floor(allPoints[i].y/blockLength)*gridDim+floor(allPoints[i].z/blockLength)*gridDim*gridDim;
        pointsToBlock[i]=blockId;
        perBlockPoints[blockId]++;

    }
}



void arrangePointsbyblock(int *pointsToBlock,struct Point* allPoints,struct Point* newPoints,int numberOfPoints,int numberOfBlocks,int* perBlockPoints,int* startingPoint){
    int k=0;
    int sum=0;
    for(int j=0;j<numberOfBlocks;j++){
        for (int i=0;i<numberOfPoints;i++) {
            if (pointsToBlock[i] == j) {
                newPoints[k] = allPoints[i];
                k++;

            }
        }
        startingPoint[j]=sum;
        sum=sum+perBlockPoints[j];
    }
}



int main(int argc,char** argv){

    int numberOfcPoints = 22;
    numberOfcPoints = pow(2, numberOfcPoints);
    int dimOfGrid = 6;
    dimOfGrid = pow(2, dimOfGrid);
    int numberOfBlocks = pow(dimOfGrid, 3);
    int numberOfqpoints = numberOfcPoints;
    struct Point *cpoints = malloc(numberOfcPoints * sizeof(struct Point));
    struct Point *qpoints = malloc(numberOfqpoints * sizeof(struct Point));

    //struct Point *knn = malloc(numberOfqpoints * sizeof( Point));
    struct Point *arrangecpoints=malloc(numberOfcPoints*sizeof( Point));
    struct Point *arrangeqpoints==malloc(numberOfqPoints*sizeof(Point));
    int *pointsctoblock = malloc(numberOfcPoints * sizeof(int));
    int *pointsqtoblock = malloc(numberOfqpoints * sizeof(int));
    int *perblockcpoints = malloc(numberOfBlocks * sizeof(int));
    int *perblockqpoints = malloc(numberOfBlocks * sizeof(int));
    int *startingpoint_c=malloc(numberOfBlocks*sizeof(int));
    int *startingpoint_q=malloc(numberOfBlocks*sizeof(int));

    generatePoints(cpoints, numberOfcPoints,1);
    generatePoints(qpoints, numberOfqpoints,2);
    //printf("-------------C  POINTS-----------\n");
    //PrintPoints(cpoints,numberOfcPoints);
    //printf("-------------Q POINTS------------\n");
    //PrintPoints(qpoints,numberOfqpoints);
    struct timeval start_t,end_t;
    gettimeofday(&start_t,NULL);
    float block_length = ((float) 1) / ((float) dimOfGrid);

    //call function for fragmentation
    putPointsBlocks(pointsctoblock, cpoints, perblockcpoints, numberOfcPoints, dimOfGrid, block_length);
    putPointsBlocks(pointsqtoblock, qpoints, perblockqpoints, numberOfqpoints, dimOfGrid, block_length,pointsqtoblockDim);
    arrangePointsbyblock(pointsctoblock,cpoints,arrangecpoints,numberOfcPoints,numberOfBlocks,perblockcpoints,startingpoint_c);
    arrangePointsbyblock(pointsqtoblock,qpoints,arrangeqpoints,numberOfqpoints,numberOfBlocks,perblockqpoints,startingpoint_q);
    free(pointsctoblock);
    free(pointsqtoblock);
    hipError_t err;
    err=hipMalloc(&arrangecpointsDev,numberOfcPoints*sizeof(Point));
    if (err != hipSuccess)                             // `hipSuccess` is provided by CUDA.
    {
        printf("Error: %s\n", hipGetErrorString(err)); // `hipGetErrorString` is provided by CUDA.
    }
    hipMemcpy=(arrangecpointsDev,arrangecpoints,numberOfcPoints*sizeof(Point),hipMemcpyHostToDevice);

    err=hipMalloc(&arrangeqpointsDev,numberOfcPoints*sizeof(Point));
    if (err != hipSuccess)                             // `hipSuccess` is provided by CUDA.
    {
        printf("Error: %s\n", hipGetErrorString(err)); // `hipGetErrorString` is provided by CUDA.
    }
    hipMemcpy=(arrangeqpointsDev,arrangeqpoints,numberOfqPoints*sizeof(Point),hipMemcpyHostToDevice);


    err=hipMalloc(&startingpointDev_c,numberOfBlocks*sizeof(int));
    if (err != hipSuccess)                             // `hipSuccess` is provided by CUDA.
    {
        printf("Error: %s\n", hipGetErrorString(err)); // `hipGetErrorString` is provided by CUDA.
    }
    hipMemcpy(startingpointDev_c,startingpoint_c,numberOfBlocks*sizeof(int),hipMemcpyHostToDevice);

    err=hipMalloc(&startingpointDev_q,numberOfBlocks*sizeof(int));
    if (err != hipSuccess)                             // `hipSuccess` is provided by CUDA.
    {
        printf("Error: %s\n", hipGetErrorString(err)); // `hipGetErrorString` is provided by CUDA.
    }
    hipMemcpy(startingpointDev_q,startingpoint_q,numberOfBlocks*sizeof(int),hipMemcpyHostToDevice);


    err=hipMalloc(&perblockcpointsDev,numberOfBlocks*sizeof(int));
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(perblockcpointsDev,perblockcpoints,numberOfBlocks*sizeof(int),hipMemcpyHostToDevice);

    err=hipMalloc(&perblockqpointsDev,numberOfBlocks*sizeof(int));
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(perblockqpointsDev,perblockqpoints,numberOfBlocks*sizeof(int),hipMemcpyHostToDevice);

    err=hipMalloc(&knn_Dev,numberOfqpoints*sizeof(Point));
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    gridDim=dim3(dimOfGrid,dimOfGrid,dimOfGrid);

    err=hipMalloc(&knnDist_Dev,numberOfqpoints*sizeof(float));
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    gridDim=dim3(dimOfGrid,dimOfGrid,dimOfGrid);

    knn<<<griDim,1024>>>(arrangecpointsDev,arrangeqpointsDev,perblockcpointsDev,perblockqpointsDev,perblockqpointsDev,startingpointDev_c,startingpointDev_q,knn_Dev,knnDist_dev)
    getimeofday(&end_t,NULL);
    par_time = (double)((end_t.tv_usec - start_t.tv_usec)/1.0e6
                        + endw_t.tv_sec - start_t.tv_sec);

    hipMemcpy(knn_Dist,knnDist_dev,numberOfqpoints*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(knn,knnDev,numberOfqpoints*sizeof(Point),hipMemcpyDeviceToHost);

    PrintKnn(qpoints,knn,numberOfqpoints);


    free(cpoints);
    free(qpoints);
    free(arrangecpoints);
    free(arrangeqpoints);
    free(perblockcpoints);
    free(perblockqpoints);
    free(startingpoint_c);
    free(startingpoint_q);
    
    hipDeviceReset();
    return 0;
}










