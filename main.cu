#include "hip/hip_runtime.h"


// Created by Filippos Kasioulis on 13/01/2019.
//

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>




struct Point {
    float x;
    float y;
    float z;

};

struct Point2{
    int x;
    int y;
    int z;
};



typedef  struct {
    int  num_of_blocks;
    int  *neighbor_blocks;

}neighborBlock;

void PrintPoints(struct Point *arr,int number){
    for (int i = 0; i < number; i++) {

        printf("Point %d: x=%f y=%f z=%f \n", i,arr[i].x, arr[i].y, arr[i].z);
    }
}

void PrintKnn(struct Point *arr,struct Point *arr2,int number ) {
    for (int i = 0; i < number; i++) {

        printf("Nearest to Point %d: x=%f y=%f z=%f--->  x=%f y=%f z=%f\n", i,arr[i].x, arr[i].y, arr[i].z,arr2[i].x, arr2[i].y, arr2[i].z);

    }
}

void generatePoints( struct Point* arr ,int number,int seed){
    srand(seed);// randomize seed
    for (int i=0;i<number;i++){
        arr[i].x=((float)rand()/(float)RAND_MAX);
        arr[i].y=((float)rand()/(float)RAND_MAX);
        arr[i].z=((float)rand()/(float)RAND_MAX);

    }

}

float distanceOfPoints( struct Point p1,struct Point p2){
    float dist=0;
    dist=pow(p1.x-p2.x,2)+pow(p1.y-p2.y,2)+pow(p1.z-p2.z,2);
    dist=sqrt(dist);
    return dist;
}


void putPointsBlocks(int *pointsToBlock,struct Point* allPoints,int* perBlockPoints,int numberOfPoints,int gridDim,float blockLength,struct Point2* pointsToBlockDim) {
    int blockId=0;
    for (int i=0; i<numberOfPoints;i++){

        blockId=floor(allPoints[i].x/blockLength)+floor(allPoints[i].y/blockLength)*gridDim+floor(allPoints[i].z/blockLength)*gridDim*gridDim;
        pointsToBlockDim[i].x=floor(allPoints[i].x/blockLength);
        pointsToBlockDim[i].y=floor(allPoints[i].y/blockLength);
        pointsToBlockDim[i].z=floor(allPoints[i].z/blockLength);
        pointsToBlock[i]=blockId;
        perBlockPoints[blockId]++;

    }
}



void putPointsBlocks2(int *pointsToBlock,struct Point* allPoints,int* perBlockPoints,int numberOfPoints,int gridDim,float blockLength) {
    int blockId=0;
    for (int i=0; i<numberOfPoints;i++){

        blockId=floor(allPoints[i].x/blockLength)+floor(allPoints[i].y/blockLength)*gridDim+floor(allPoints[i].z/blockLength)*gridDim*gridDim;
        pointsToBlock[i]=blockId;
        perBlockPoints[blockId]++;

    }

}


void arrangePointsbyblock(int *pointsToBlock,struct Point* allPoints,struct Point* newPoints,int numberOfPoints,int numberOfBlocks,int* perBlockPoints,int* startingPoint){
    int k=0;
    int sum=0;
    for(int j=0;j<numberOfBlocks;j++){
        for (int i=0;i<numberOfPoints;i++) {
            if (pointsToBlock[i] == j) {
                newPoints[k] = allPoints[i];
                k++;


            }
        }
        startingPoint[j]=sum;
        sum=sum+perBlockPoints[j];
    }
}




void getNeighbourBlocks(int gridDim,struct Point2 blockDim,neighborBlock* neighbor) {
    int *temp = malloc( 27 * sizeof(int));//27 the max number of neighbours
    int size=0;

    for (int i = -1; i < 2; i++) {
        for (int j = -1; j < 2; j++) {
            for (int k = -1; k < 2; k++) {
                if (!(i == 0 & j == 0 & k == 0)) {
                    int tempx = blockDim.x+i;
                    int tempy = blockDim.y+j;
                    int tempz = blockDim.z+k;
                    if (!(tempx < 0 | tempy < 0 | tempz < 0 | tempx >= gridDim | tempz >= gridDim | tempy >= gridDim)) {
                        temp[size]=tempx+tempy*gridDim+tempz*gridDim*gridDim;
                        size++;
                    }
                }
            }
        }
    }
    (*neighbor).num_of_blocks=size;
    //printf("%d %d %d \n ",blockDim.x,blockDim.y,blockDim.z);
    (*neighbor).neighbor_blocks = malloc(size*sizeof(int));
    memcpy(neighbor->neighbor_blocks , temp , size*sizeof(int));
    free(temp);
}

struct  Point searchMinDistanceblock(neighborBlock* neig,struct Point* allPoints,int* startingpoint_c,int* perblockcpoints,struct Point pointtosearch  ){
    float minDist=999;
    float tempDist;
    struct Point minCand;
    int blockId;
    //printf("number of blocks %d\n",neig->num_of_blocks);
    for (int j=0;j<neig->num_of_blocks;j++){
        blockId=neig->neighbor_blocks[j];
        for (int i = startingpoint_c[blockId]; (i < (startingpoint_c[blockId]+perblockcpoints[blockId])); i++) {

            //printf("blocks %d",neig->neighbor_blocks[j]);
            tempDist=distanceOfPoints(pointtosearch,allPoints[i]);
            if (tempDist<minDist){
                minDist=tempDist;
                minCand=allPoints[i];
            }


        }
    }
    //printf("mincand from neighbors %f %f %f\n",minCand.x,minCand.y,minCand.z);
    free(neig->neighbor_blocks);
    return minCand;

}

void validation(struct Point* knn,struct Point* qpoints,struct  Point* cpoints,int numberPoints){
    float success=0;
    float tempDist;
    float minDist;
    struct Point minCordCan;
    for (int i=0;i<numberPoints;i++){
        tempDist=0;
        minDist=99;
        for(int j=0;j<numberPoints;j++) {
            tempDist = distanceOfPoints(qpoints[i], cpoints[j]);
            if (tempDist < minDist) {
                minDist = tempDist;
                minCordCan.x = cpoints[j].x;
                minCordCan.y=cpoints[j].y;
                minCordCan.z=cpoints[j].z;

            }
        }

        if ((minCordCan.x==knn[i].x)&(minCordCan.y==knn[i].y)&(minCordCan.z==knn[i].z))
        {
            printf("ok %d %f %f %f \n ",i,minCordCan.x,minCordCan.y,minCordCan.z);
            success++;
        }
        else
            printf("not ok %d %f %f %f \n ",i,minCordCan.x,minCordCan.y,minCordCan.z);
    }

    printf("Success ratee %.1f\n",(success/numberPoints)*100);

}



int main(int argc,char** argv) {

    struct timeval start_t,end_t;
    double ser_time;
    gettimeofday(&start_t,NULL);
    int numberOfcPoints = 18;
    numberOfcPoints = pow(2, numberOfcPoints);
    int dimOfGrid = 4;
    dimOfGrid = pow(2, dimOfGrid);
    int numberOfBlocks = pow(dimOfGrid, 3);
    int numberOfqpoints = numberOfcPoints;
    struct Point *cpoints = malloc(numberOfcPoints * sizeof(struct Point));
    struct Point *qpoints = malloc(numberOfqpoints * sizeof(struct Point));
    struct Point2 *pointsqtoblockDim = malloc(numberOfqpoints * sizeof(struct Point2));
    struct Point2 *pointsctoblockDim = malloc(numberOfcPoints * sizeof(struct Point2));

    struct Point *knn = malloc(numberOfqpoints * sizeof(struct Point));
    struct Point *arrangecpoints=malloc(numberOfcPoints*sizeof(struct Point));
    //struct Point *arrangeqpoints=malloc(numberOfqpoints*sizeof(struct Point));
    int *pointsctoblock = malloc(numberOfcPoints * sizeof(int));
    int *pointsqtoblock = malloc(numberOfqpoints * sizeof(int));
    int *perblockcpoints = malloc(numberOfBlocks * sizeof(int));
    int *perblockqpoints = malloc(numberOfBlocks * sizeof(int));
    int *startingpoint_c=malloc(numberOfBlocks*sizeof(int));
    //int *startingpoint_q=malloc(numberOfBlocks* sizeof(int));



    generatePoints(cpoints, numberOfcPoints,1);
    generatePoints(qpoints, numberOfqpoints,2);
    //printf("-------------C  POINTS-----------\n");
    PrintPoints(cpoints,numberOfcPoints);
    //printf("-------------Q POINTS------------\n");
    PrintPoints(qpoints,numberOfqpoints);

    float block_length = ((float) 1) / ((float) dimOfGrid);

//call function for fragmentation
    putPointsBlocks2(pointsctoblock, cpoints, perblockcpoints, numberOfcPoints, dimOfGrid, block_length);
    putPointsBlocks(pointsqtoblock, qpoints, perblockqpoints, numberOfqpoints, dimOfGrid, block_length,pointsqtoblockDim);
    arrangePointsbyblock(pointsctoblock,cpoints,arrangecpoints,numberOfcPoints,numberOfBlocks,perblockcpoints,startingpoint_c);
    //arrangePointsbyblock(pointsqtoblock,qpoints,arrangeqpoints,numberOfqpoints,numberOfBlocks,perblockcpoints,startingpoint_q);


//find the primary candidates of each queries
    for (int q = 0; q < numberOfqpoints; q++) {
        int blockId;
        struct Point2 blockDim;
        blockId=pointsqtoblock[q];
        blockDim.x=pointsqtoblockDim[q].x;
        blockDim.y=pointsqtoblockDim[q].y;
        blockDim.z=pointsqtoblockDim[q].z;
        //printf(" %d %d %d  \n",blockDim.x,blockDim.y,blockDim.z);

        //printf(" %d %d %d  \n",blockDim.x,blockDim.y,blockDim.z);
        //struct Point *primaryCandidates = malloc(perblockcpoints[blockId] * sizeof(struct Point));
        int k = 0;


        float tempDistance;
        struct Point minCandidateCord;
        float minDistCand = 999;
        for (int i = startingpoint_c[blockId]; (i < (startingpoint_c[blockId]+perblockcpoints[blockId])); i++) {
            tempDistance = distanceOfPoints(qpoints[q],arrangecpoints[i]);
            if (tempDistance < minDistCand) {
                minDistCand = tempDistance;
                minCandidateCord= arrangecpoints[i];

            }
        }
        //printf("Point %d minCandidateCord  %f %f %f\n" ,q,minCandidateCord.x,minCandidateCord.y,minCandidateCord.z);
        float min_from_bounds = 999;
        float tempDistancex = qpoints[q].x-(block_length * blockDim.x);
        if ((tempDistancex > block_length - tempDistancex)&&(blockDim.x<dimOfGrid-1))
            tempDistancex = block_length - tempDistancex;
        min_from_bounds = tempDistancex;
        float tempDistancey = qpoints[q].y-(block_length * blockDim.y);
        if ((tempDistancey > block_length - tempDistancey)&&(blockDim.y<dimOfGrid-1))
            tempDistancey = block_length - tempDistancey;
        if (tempDistancey < min_from_bounds)
            min_from_bounds = tempDistancey;


        float tempDistancez = qpoints[q].z-(block_length * blockDim.z);
        if ((tempDistancez > block_length - tempDistancez)&&(blockDim.z<dimOfGrid-1))
            tempDistancez = block_length - tempDistancez;
        if (tempDistancez < min_from_bounds)
            min_from_bounds = tempDistancez;


        if (min_from_bounds < minDistCand) {
            printf("%d\n",q);
            neighborBlock neighbour;
            getNeighbourBlocks(dimOfGrid, blockDim, &neighbour);
            //printf("number of neighbors %d\n", neighbour.num_of_blocks);
            struct Point minCordNeig;
            minCordNeig = searchMinDistanceblock(&neighbour,arrangecpoints,startingpoint_c,perblockcpoints,qpoints[q]);
            if (distanceOfPoints(minCordNeig, qpoints[q]) < minDistCand)
                minCandidateCord = minCordNeig;

        }
        knn[q] = minCandidateCord;


    }


    //printf("--------------------------------------K-nn---------------------------------\n");
    //PrintKnn(qpoints, knn, numberOfqpoints);
    gettimeofday(&end_t,NULL);
    ser_time = (double)((end_t.tv_usec - start_t.tv_usec)/1.0e6
                           + endw_t.tv_sec - start_t.tv_sec);
    printf("Serial time of calculation :%f sec\n",ser_time)
    //validation(knn,qpoints,cpoints,numberOfqpoints);
    free(cpoints);
    free(qpoints);
    free(perblockcpoints);
    free(perblockqpoints);
    free(pointsqtoblockDim);
    free(pointsctoblockDim);
    free(knn);
    free(pointsqtoblock);
    free(pointsctoblock);
    free(startingpoint_c);
    free(arrangecpoints);
    return 0;

}